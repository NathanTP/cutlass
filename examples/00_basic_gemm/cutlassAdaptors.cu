//#include "helper.h"
#include "cutlassAdaptors.h"

extern "C"
CutlassGemm::GemmKernel::Params *adaptSGEMMArgs(
  int M,
  int N,
  int K,
  float alpha,
  float const *A,
  int lda,
  float const *B,
  int ldb,
  float beta,
  float *C,
  int ldc) {
  // Define a CUTLASS GEMM type
  CutlassGemm gemm_operator;

  CutlassGemm::Arguments args({M , N, K},  // Gemm Problem dimensions
                              {A, lda},    // Tensor-ref for source matrix A
                              {B, ldb},    // Tensor-ref for source matrix B
                              {C, ldc},    // Tensor-ref for source matrix C
                              {C, ldc},    // Tensor-ref for destination matrix D (may be different memory than source C matrix)
                              {alpha, beta}); // Scalars used in the Epilogue

  // Launch the CUTLASS GEMM kernel.
  hipStream_t stream = nullptr;
  gemm_operator.initialize(args, stream=stream);
  CutlassGemm::GemmKernel::Params params_ = gemm_operator.get_params();
  CutlassGemm::GemmKernel::Params *params_ptr = (CutlassGemm::GemmKernel::Params*) malloc(sizeof(params_));
  memcpy(params_ptr, &params_, sizeof(params_));
  return params_ptr;
}

extern "C"
CudaConfig *getCudaConfig(CutlassGemm::GemmKernel::Params *params_ptr) {
  CutlassGemm::ThreadblockSwizzle threadblock_swizzle;
  dim3 grid = threadblock_swizzle.get_grid_shape(params_ptr->grid_tiled_shape);
  dim3 block(CutlassGemm::GemmKernel::kThreadCount, 1, 1);
  int smem_size = int(sizeof(typename CutlassGemm::GemmKernel::SharedStorage));
  CudaConfig* ptr = new CudaConfig;
  ptr->grid_ = grid;
  ptr->block_ = block;
  ptr->smem_size_ = smem_size;
  return ptr;
}